#include "Board.h"
#include "Body.h"
#include "Creature.h"
#include "Genome.h"
#include "device_util.cuh"
#include "host_util.h"
#include "testing/test.h"
#include "util.h"
#include <algorithm>
#include <stack>

int main()
{
    read_in_settings();
    print(BOARD_WIDTH, BOARD_HEIGHT);
    // initialize board
    Board* board;
    hipMallocManaged((void**)&board, sizeof(Board));
    int* grass;
    int* grass_stage;
    hipMallocManaged((void**)&grass, sizeof(int) * BOARD_WIDTH * BOARD_HEIGHT);
    hipMallocManaged((void**)&grass_stage, sizeof(int) * BOARD_WIDTH * BOARD_HEIGHT);
    *board = Board(grass, grass_stage);
    int id_gen = 0;
    // int MAX_CREATURES = 10000;
    stack<int> available_idx;

    // initialize bodies
    int num_creatures = INIT_CREATURES;
    Body* bodies;
    hipMallocManaged((void**)&bodies, sizeof(Body) * MAX_CREATURES);
    vector<Creature> creatures;
    for (int i = 0; i < num_creatures; i++) {
        int x = random_int(0, BOARD_WIDTH - 1);
        int y = random_int(0, BOARD_HEIGHT - 1);
        Genome g(random_int(WEIGHTS, 1000));
        creatures.emplace_back(Creature(x, y, -1, -1, id_gen, 0, bodies + i, g));
        id_gen++;
    }
    for (int i = MAX_CREATURES - 1; i >= num_creatures; i--) {
        bodies[i].m_alive = false;
        available_idx.push(i);
    }
    ll n = 0;
    dim3 grass_block(32, 32);
    dim3 grass_grid((BOARD_WIDTH + grass_block.x - 1) / grass_block.x, (BOARD_HEIGHT + grass_block.y - 1) / grass_block.y);
    // run a few cycles
    while (true) {
        using namespace chrono;
        auto g_start = high_resolution_clock::now();
        grow_grass<<<grass_grid, grass_block>>>(board, GRASS_MAX_HEIGHT, GRASS_PERIOD, BOARD_WIDTH, BOARD_HEIGHT);
        hipDeviceSynchronize();
        int dimx = 1024;
        dim3 block(dimx);
        dim3 grid((MAX_CREATURES + block.x - 1) / block.x);
        get_inputs<<<grid, block>>>(board, bodies, MAX_CREATURES, BOARD_WIDTH, BOARD_HEIGHT);
        hipDeviceSynchronize();
        think_and_act<<<grid, block>>>(bodies, MAX_CREATURES, COST_MOVEMENT, COEFF_BASE_ENERGY, STARVATION, HEALTH_TO_ENERGY_RATIO, ENERGY_TO_HEALTH, GRASS_MAX_HEIGHT, GRASS_PERIOD, BOARD_WIDTH, BOARD_HEIGHT);
        hipDeviceSynchronize();
        auto g_end = high_resolution_clock::now();
        remove_dead(creatures, available_idx);
        eat_and_attack(board, bodies, MAX_CREATURES);
        remove_dead(creatures, available_idx);
        auto start = high_resolution_clock::now();
        birth(creatures, bodies, MAX_CREATURES, id_gen, available_idx);
        auto end = high_resolution_clock::now();
        remove_dead(creatures, available_idx);

        vector<double> x_vec;
        vector<double> y_vec;
        vector<int> c_vec;
        num_creatures = 0;
        int total_mass = 0;
        int total_energy = 0;
        int total_preds = 0;
        int total_preys = 0;
        int total_brain_size = 0;
        for (int i = 0; i < MAX_CREATURES; i++) {
            if (bodies[i].m_alive) {
                x_vec.pb(bodies[i].m_x);
                y_vec.pb(bodies[i].m_y);
                if (bodies[i].m_output_values[ATTACK] > 0) {
                    total_preds++;
                    c_vec.pb(1);
                } else {
                    total_preys++;
                    c_vec.pb(0);
                }
                // dprint(bodies[i].m_health, bodies[i].m_energy, bodies[i].m_base_energy_use, bodies[i].m_output_values[ATTACK]);
                num_creatures++;
                total_mass += bodies[i].m_health;
                total_energy += bodies[i].m_energy;
            }
            if (i < (int)creatures.size() && creatures[i].m_body->m_alive) {
                total_brain_size += creatures[i].m_genome.m_bitstring.size();
            }
        }
        // vector<int> grass_x;
        // vector<int> grass_y;
        // vector<int> grass_c;
        // for (int i = 0; i < BOARD_WIDTH; i++) {
        //     for (int j = 0; j < BOARD_HEIGHT; j++) {
        //         grass_x.pb(i);
        //         grass_y.pb(j);
        //         grass_c.pb(board->grass[i][j]);
        //     }
        // }
        print(n);
        print(total_mass);
        print(total_energy);
        print(total_preds);
        print(total_preys);
        print(total_brain_size);
        dprint(num_creatures);
        dprint("GPU took", duration_cast<milliseconds>(g_end - g_start).count());
        dprint("CPU took", duration_cast<milliseconds>(end - start).count());
        // print(grass_x);
        // print(grass_y);
        // print(grass_c);
        print(x_vec);
        print(y_vec);
        print(c_vec);
        n++;
        // cerr << n << endl;
        // sleep(1);
    }
}