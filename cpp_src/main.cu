#include "Board.h"
#include "Body.h"
#include "Creature.h"
#include "Genome.h"
#include "device_util.cuh"
#include "testing/test.h"
#include "util.h"
#include <algorithm>

int main()
{
    // initialize board
    Board* board;
    hipMallocManaged((void**)&board, sizeof(Board));
    *board = Board();

    // initialize bodies
    int num_creatures = 100;
    Body* bodies;
    hipMallocManaged((void**)&bodies, sizeof(Body) * num_creatures);
    vector<Creature> creatures;
    for (int i = 0; i < num_creatures; i++) {
        int x = random_int(0, BOARD_WIDTH - 1);
        int y = random_int(0, BOARD_HEIGHT - 1);
        Genome g(1000);
        creatures.emplace_back(Creature(x, y, 100, i, 0, bodies + i, g));
    }
    ll n = 1000;
    dim3 grass_block(32, 32);
    dim3 grass_grid((BOARD_WIDTH + grass_block.x - 1) / grass_block.x, (BOARD_HEIGHT + grass_block.y - 1) / grass_block.y);
    // run a few cycles
    while (true) {
        grow_grass<<<grass_grid, grass_block>>>(board);
        hipDeviceSynchronize();
        int dimx = 32;
        dim3 block(dimx);
        dim3 grid((num_creatures + block.x - 1) / block.x);
        get_inputs<<<grid, block>>>(board, bodies, num_creatures);
        hipDeviceSynchronize();
        think_and_act<<<grid, block>>>(bodies, num_creatures);
        hipDeviceSynchronize();

        vector<double> x_vec;
        vector<double> y_vec;
        for (int i = 0; i < num_creatures; i++) {
            x_vec.pb(bodies[i].m_x);
            y_vec.pb(bodies[i].m_y);
        }
        print(x_vec);
        print(y_vec);
        // print(-1);
        // cerr << n << endl;
        // sleep(1);
    }
}