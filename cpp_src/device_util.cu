#include "hip/hip_runtime.h"
#include "Board.h"
#include "Body.h"
#include "device_util.cuh"
#include "util.h"
#include <stdio.h>

__device__ int get_delta(int t, int s, int axis)
{
    int d = t - s;
    if (d > (axis / 2)) {
        d -= axis;
    } else if (d < (-axis / 2)) {
        d += axis;
    }
    return d;
}
// puts input vectors in-place into bodies
__global__ void get_inputs(Board* board, Body* bodies, int num_bodies)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;

    if (ix < num_bodies && bodies[ix].m_alive) {
        int x = bodies[ix].m_x;
        int y = bodies[ix].m_y;
        // find closest grass
        int dist = BOARD_WIDTH + BOARD_HEIGHT;
        for (int i = 0; i < BOARD_WIDTH; i++) {
            for (int j = 0; j < BOARD_HEIGHT; j++) {
                if (board->grass[i][j]) {
                    int dx = get_delta(i, x, BOARD_WIDTH);
                    int dy = get_delta(j, y, BOARD_HEIGHT);
                    int cand_dist = abs(dx) + abs(dy);
                    if (cand_dist < dist) {
                        dist = cand_dist;
                        bodies[ix].m_sensory_input[NEAREST_GRASS_X] = dx;
                        bodies[ix].m_sensory_input[NEAREST_GRASS_X] = dy;
                    }
                }
            }
        }
        // find closest neighbor
        dist = BOARD_WIDTH + BOARD_HEIGHT;
        for (int i = 0; i < num_bodies; i++) {
            if (i == ix)
                continue;
            int t_i = bodies[i].m_x;
            int t_j = bodies[i].m_y;
            int dx = get_delta(t_i, x, BOARD_WIDTH);
            int dy = get_delta(t_j, y, BOARD_WIDTH);
            int cand_dist = abs(dx) + abs(dy);
            if (cand_dist < dist) {
                dist = cand_dist;
                bodies[ix].m_sensory_input[NEAREST_CREATURE_X] = dx;
                bodies[ix].m_sensory_input[NEAREST_CREATURE_Y] = dy;
                bodies[ix].m_sensory_input[NEAREST_CREATURE_HEALTH] = bodies[i].m_health;
            }
        }
        bodies[ix].m_sensory_input[HEALTH] = bodies[ix].m_health;
        bodies[ix].m_sensory_input[ENERGY] = bodies[ix].m_energy;
        bodies[ix].m_sensory_input[BIAS] = 1;
        // Random will be handled by the host
    }
}

// Does the matrix multiplication, then updates positions
__global__ void think_and_act(Body* bodies, int num_bodies)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    if (ix < num_bodies && bodies[ix].m_alive) {
        Body& body = bodies[ix];
        double* actions = body.m_output_values;
        for (int i = 0; i < SIZE_HIDDEN_LAYER; i++) {
            for (int j = 0; j < SIZE_INPUT_LAYER; j++) {
                body.m_hidden_values[i] += body.m_hidden_weights[i][j] * body.m_sensory_input[j];
            }
            body.m_hidden_values[i] = atan(body.m_hidden_values[i]);
        }
        for (int i = 0; i < SIZE_OUTPUT_LAYER; i++) {
            for (int j = 0; j < SIZE_INPUT_LAYER + SIZE_HIDDEN_LAYER; j++) {
                body.m_output_values[i] += body.m_output_weights[i][j] * body.m_sensory_input[j];
            }
            body.m_output_values[i] = atan(body.m_output_values[i]);
        }
        // always allow a move
        body.m_x += actions[MOVE_X] * body.m_speed;
        body.m_x = (body.m_x + BOARD_WIDTH) % BOARD_WIDTH;
        body.m_y += actions[MOVE_Y] * body.m_speed;
        body.m_y = (body.m_y + BOARD_HEIGHT) % BOARD_HEIGHT;
        body.m_energy -= (abs(actions[MOVE_X]) + abs(actions[MOVE_Y])) * COST_MOVEMENT;
        body.m_energy -= ceil(body.m_base_energy_use * COEFF_BASE_ENERGY);
        if (body.m_energy < 0) {
            body.m_health -= STARVATION;
            body.m_energy = STARVATION * HEALTH_TO_ENERGY_RATIO * ENERGY_TO_HEALTH;
        }
        // // regenerate
        // else if (body.m_energy >= ENERGY_TO_HEALTH) {
        //     body.m_energy -= ENERGY_TO_HEALTH;
        //     body.m_health = min(body.m_health + 1, body.m_max_health);
        // }
        // grass eating, attacking, and splitting are handled host-side
    }
}

// Grows grass!
__global__ void grow_grass(Board* board)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    if (ix < BOARD_WIDTH && iy < BOARD_HEIGHT && board->grass[ix][iy] < GRASS_MAX_HEIGHT) {
        board->grass_stage[ix][iy]++;
        if (board->grass_stage[ix][iy] == GRASS_PERIOD) {
            board->grass_stage[ix][iy] = 0;
            board->grass[ix][iy]++;
            board->grass[ix][iy] = min(board->grass[ix][iy], GRASS_MAX_HEIGHT);
        }
    }
}